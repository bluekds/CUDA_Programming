﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

#define _1MB (1024*1024)

void main(void) {
    int ngpus;
    hipGetDeviceCount(&ngpus);

    for (int i = 0; i < ngpus; i++) {
        hipDeviceProp_t devProp;

        hipGetDeviceProperties(&devProp, i);

        printf("Device %d: %s\n"
            , i, devProp.name);
        printf("\tCompute capability: %d.%d\n"
            , devProp.major, devProp.minor);
        printf("\tThe number of streaming multiprocessors: %d\n"
            , devProp.multiProcessorCount);
        printf("\tThe number of CUDA cores: %d\n"
            , _ConvertSMVer2Cores(devProp.major, devProp.minor)
              * devProp.multiProcessorCount);
        printf("\tGlobal memory size: %.2f MB"
            , (float)devProp.totalGlobalMem / _1MB);
    }
}
