#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define DATA_TYPE int

#define SIZE_M (512*2)
#define SIZE_N (512*4)
#define SIZE_K (512*2)

#define INDEX2ROW(_index,_width)	(int)((_index)/(_width))
#define INDEX2COL(_index,_width)	((_index)%(_width))
#define ID2INDEX(_row,_col, _width) (((_row)*(_width))+(_col))

#define BLOCK_SIZE 16

// macro function
#define IS_EQUAL(_a, _b) (abs(_b - _a) < 10e-6)

/******************************************************************
* Modify this kernel to use shared memory
******************************************************************/
__global__ void MatMul_SharedMem(DATA_TYPE* matA, DATA_TYPE* matB, DATA_TYPE* matC, int m, int n, int k)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	int col = blockDim.y * blockIdx.y + threadIdx.y;

	DATA_TYPE val = 0;
	__shared__ DATA_TYPE subA[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ DATA_TYPE subB[BLOCK_SIZE][BLOCK_SIZE];

	int localRow = threadIdx.x;
	int localCol = threadIdx.y;

	for (int bID = 0; bID < ceil((float)k / BLOCK_SIZE); bID++) {
		int offset = bID * BLOCK_SIZE;

		// load A and B
		if (row >= m || offset + localCol >= k)
			subA[localRow][localCol] = 0;
		else
			subA[localRow][localCol] = matA[row * k + (offset + localCol)];

		if (col >= n || offset + localRow >= k)
			subB[localRow][localCol] = 0;
		else
			subB[localRow][localCol] = matB[(offset + localRow) * n + col];

		__syncthreads();

		// compute
		for (int i = 0; i < BLOCK_SIZE; i++) {
			val += subA[localRow][i] * subB[i][localCol];
		}
		__syncthreads();
	}

	if (row >= m || col >= n)
		return;

	matC[row * n + col] = val;
}
/******************************************************************
******************************************************************/

template<class T> void allocNinitMem(T** p, long long size, DATA_TYPE* memUsage = NULL);
void runMatMul_Basic(DATA_TYPE* matA, DATA_TYPE* matB, DATA_TYPE* matC, int m, int n, int k);
bool compareMatrix(DATA_TYPE* _A, DATA_TYPE* _B, int _size);

DS_timer timer(10);
void setTimer();

int main(int argc, char* argv[])
{
	setTimer();

	// set matrix size
	int m, n, k;
	m = SIZE_M;
	n = SIZE_N;
	k = SIZE_K;

	printf("Size : A = (%d by %d), B = (%d by %d), C = (%d by %d)\n", m, k, k, n, m, n);

	int sizeA = m * k;
	int sizeB = k * n;
	int sizeC = m * n;

	// Make matrix
	DATA_TYPE* A = NULL, * B = NULL;
	allocNinitMem<DATA_TYPE>(&A, sizeA);
	allocNinitMem<DATA_TYPE>(&B, sizeB);

	DATA_TYPE* Ccpu = NULL, * Cgpu = NULL;
	allocNinitMem<DATA_TYPE>(&Ccpu, sizeC);
	allocNinitMem<DATA_TYPE>(&Cgpu, sizeC);

	// generate input matrices
	for (int i = 0; i < sizeA; i++) A[i] = ((rand() % 10) + ((rand() % 100) / 100.0));
	for (int i = 0; i < sizeB; i++) B[i] = ((rand() % 10) + ((rand() % 100) / 100.0));

	// CPU version (OpenMP)
	timer.onTimer(0);
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			int cIndex = row * n + col;
			Ccpu[cIndex] = 0;
			for (int i = 0; i < k; i++)
				Ccpu[cIndex] += (A[row * k + i] * B[i * n + col]);
		}
	}
	printf("CPU finished!\n");
	timer.offTimer(0);

	// GPU setup
	DATA_TYPE* dA, * dB, * dC;

	hipMalloc(&dA, sizeA * sizeof(DATA_TYPE));
	hipMemset(dA, 0, sizeA * sizeof(DATA_TYPE));

	hipMalloc(&dB, sizeB * sizeof(DATA_TYPE));
	hipMemset(dB, 0, sizeB * sizeof(DATA_TYPE));

	hipMalloc(&dC, sizeC * sizeof(DATA_TYPE));
	hipMemset(dC, 0, sizeC * sizeof(DATA_TYPE));

	timer.onTimer(1);

	timer.onTimer(4);
	hipMemcpy(dA, A, sizeA * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeB * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	timer.offTimer(4);

	/******************************************************************
	* Write your codes for GPU algorithm from here
	******************************************************************/
	// Sharead memroy version

	// 1. set the thread layout
	// Change the layout if you need
	dim3 gridDim(ceil((float)m / BLOCK_SIZE), ceil((float)n / BLOCK_SIZE));
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
	printf("Grid(%d, %d), Block(%d, %d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

	// 2. kernel call
	timer.onTimer(3);
	MatMul_SharedMem <<<gridDim, blockDim >>> (dA, dB, dC, m, n, k);
	hipDeviceSynchronize();
	timer.offTimer(3);

	/******************************************************************
	******************************************************************/

	timer.onTimer(5);
	hipMemcpy(Cgpu, dC, sizeC * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
	timer.offTimer(5);

	timer.offTimer(1);

	// Basci version
	runMatMul_Basic(dA, dB, dC, m, n, k);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	printf("[Kernel (shared memroy)] ");
	compareMatrix(Ccpu, Cgpu, sizeC);

	timer.printTimer(1);

	delete A;
	delete B;
	delete Ccpu;
	delete Cgpu;

	return 0;
}

bool compareMatrix(DATA_TYPE* _A, DATA_TYPE* _B, int _size)
{
	bool isMatched = true;
	for (int i = 0; i < _size; i++) {
		if (!IS_EQUAL(_A[i], _B[i])) {
			printf("[%d] not matched! (%f, %f)\n", i, _A[i], _B[i]);
			getchar();
			isMatched = false;
		}
	}
	if (isMatched)
		printf("Results are matched!\n");
	else
		printf("Results are not matched!!!!!!!!!!!\n");

	return isMatched;
}

__global__ void MatMul(DATA_TYPE* matA, DATA_TYPE* matB, DATA_TYPE* matC, int m, int n, int k)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	int col = blockDim.y * blockIdx.y + threadIdx.y;

	if (row >= m || col >= n)
		return;

	DATA_TYPE val = 0; // hope to use register
	for (int i = 0; i < k; i++)
		val += matA[ID2INDEX(row, i, k)] * matB[ID2INDEX(i, col, n)];

	matC[ID2INDEX(row, col, n)] = val;
}

void runMatMul_Basic(DATA_TYPE* matA, DATA_TYPE* matB, DATA_TYPE* matC, int m, int n, int k)
{
	dim3 gridDim(ceil((float)m / BLOCK_SIZE), ceil((float)n / BLOCK_SIZE));
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

	timer.onTimer(7);
	MatMul <<< gridDim, blockDim >>> (matA, matB, matC, m, n, k);
	hipDeviceSynchronize();
	timer.offTimer(7);

	hipMemset(matC, 0, m * n * sizeof(DATA_TYPE));
}

template<class T>
void allocNinitMem(T** p, long long size, DATA_TYPE* memUsage) {
	*p = new T[size];
	memset(*p, 0, sizeof(T) * size);

	if (memUsage != NULL) {
		*memUsage += sizeof(T) * size;
	}
}

void setTimer()
{
	timer.setTimerName(0, (char*)"CPU algorithm");
	timer.setTimerName(1, (char*)"GPU/CUDA algorithm");
	timer.setTimerName(3, (char*)" - Kernel (Shared memory)");
	timer.setTimerName(4, (char*)" - [Data transfer] host->device");
	timer.setTimerName(5, (char*)" - [Data transfer] device->host");
	timer.setTimerName(7, (char*)"Kernel (Basic)");
}