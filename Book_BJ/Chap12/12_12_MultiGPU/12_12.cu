#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

void main(void) {
	int ngpus;
	hipGetDeviceCount(&ngpus);

	for (int i = 0; i < ngpus; i++) {
		hipDeviceProp_t devProp;

		hipGetDeviceProperties(&devProp, i);
		printf("Device[%d](%s) compute capability : %d.%d.\n"
			, i, devProp.name, devProp.major, devProp.minor);
	}
}