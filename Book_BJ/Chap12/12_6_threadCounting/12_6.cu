#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_BLOCK 10240
#define NUM_T_IN_B 512

__global__ void threadCounting_noSync(int* a)
{
	(*a)++;
}

__global__ void threadCounting_atomicGlobal(int* a)
{
	atomicAdd(a, 1);
}

__global__ void threadCounting_atomicShared(int* a)
{
	__shared__ int sa;

	if (threadIdx.x == 0)
		sa = 0;
	__syncthreads();

	atomicAdd(&sa, 1);
	__syncthreads();

	if (threadIdx.x == 0)
		atomicAdd(a, sa);
}

__global__ void threadCounting_warpLvSync(int* a)
{
	__shared__ int wa[NUM_T_IN_B / 32];
	__shared__ int sa;

	int warpID = (int)threadIdx.x / 32;

	if (threadIdx.x == 0)
		sa = 0;
	__syncthreads();

	if (threadIdx.x % 32 == 0)
		wa[warpID] = 0;
	__syncwarp();

	atomicAdd(&wa[warpID], 1);

	__syncwarp();

	if (threadIdx.x % 32 == 0)
		atomicAdd(&sa, wa[warpID]);

	__syncthreads();

	if (threadIdx.x == 0)
		atomicAdd(a, sa);
}

int main(void) {
	DS_timer timer(10);
	timer.setTimerName(0, (char*)"No Sync.");
	timer.setTimerName(1, (char*)"AtomicGlobal");
	timer.setTimerName(2, (char*)"AtomicShared");
	timer.setTimerName(3, (char*)"AtomicWarp");

	int a = 0;
	int* d1, * d2, * d3, *d4;

	//hipSetDevice(1);

	hipMalloc((void**)&d1, sizeof(int));
	hipMemset(d1, 0, sizeof(int) * 0);

	hipMalloc((void**)&d2, sizeof(int));
	hipMemset(d2, 0, sizeof(int) * 0);

	hipMalloc((void**)&d3, sizeof(int));
	hipMemset(d3, 0, sizeof(int) * 0);

	hipMalloc((void**)&d4, sizeof(int));
	hipMemset(d3, 0, sizeof(int) * 0);

	// warp-up
	threadCounting_noSync << <NUM_BLOCK, NUM_T_IN_B >> > (d1);
	hipDeviceSynchronize();

	timer.onTimer(0);
	threadCounting_noSync << <NUM_BLOCK, NUM_T_IN_B >> > (d1);
	hipDeviceSynchronize();
	timer.offTimer(0);

	hipMemcpy(&a, d1, sizeof(int), hipMemcpyDeviceToHost);
	printf("[No Sync.] # of threads = %d\n", a);

	timer.onTimer(1);
	threadCounting_atomicGlobal << <NUM_BLOCK, NUM_T_IN_B >> > (d2);
	hipDeviceSynchronize();
	timer.offTimer(1);

	hipMemcpy(&a, d2, sizeof(int), hipMemcpyDeviceToHost);
	printf("[AtomicGlobal] # of threads = %d\n", a);

	timer.onTimer(2);
	threadCounting_atomicShared << <NUM_BLOCK, NUM_T_IN_B >> > (d3);
	hipDeviceSynchronize();
	timer.offTimer(2);

	hipMemcpy(&a, d3, sizeof(int), hipMemcpyDeviceToHost);
	printf("[AtomicShared] # of threads = %d\n", a);

	timer.onTimer(3);
	threadCounting_warpLvSync << <NUM_BLOCK, NUM_T_IN_B >> > (d4);
	hipDeviceSynchronize();
	timer.offTimer(3);

	hipMemcpy(&a, d4, sizeof(int), hipMemcpyDeviceToHost);
	printf("[AtomicWarp] # of threads = %d\n", a);

	hipFree(d1);
	hipFree(d2);
	hipFree(d3);
	hipFree(d4);

	timer.printTimer();

	return 0;
}