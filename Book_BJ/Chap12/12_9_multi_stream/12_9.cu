#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_BLOCK (128*1024)
#define ARRAY_SIZE (1024*NUM_BLOCK)

#define NUM_STREAMS 2

#define WORK_LOAD 256

__global__ void myKernel(int* _in, int* _out)
{
	int tID = blockDim.x * blockIdx.x + threadIdx.x;

	int temp = 0;
	int in = _in[tID];
	for (int i = 0; i < WORK_LOAD; i++) {
		temp = (temp + in * 5) % 10;
	}
	_out[tID] = temp;

}

void main(void)
{
	DS_timer timer(10);
	timer.setTimerName(0, "Single stream");
	timer.setTimerName(1, "  * Host -> Device");
	timer.setTimerName(2, "  * Kernel execution");
	timer.setTimerName(3, "  * Device -> Host");
	timer.setTimerName(4, "Multiple streams");

	int* in = NULL, * out = NULL, * out2 = NULL;

	hipHostMalloc(&in, sizeof(int) * ARRAY_SIZE);
	memset(in, 0, sizeof(int) * ARRAY_SIZE);

	hipHostMalloc(&out, sizeof(int) * ARRAY_SIZE);
	memset(out, 0, sizeof(int) * ARRAY_SIZE);

	hipHostMalloc(&out2, sizeof(int) * ARRAY_SIZE);
	memset(out2, 0, sizeof(int) * ARRAY_SIZE);

	int* dIn, * dOut;
	hipMalloc(&dIn, sizeof(int) * ARRAY_SIZE);
	hipMalloc(&dOut, sizeof(int) * ARRAY_SIZE);

	for (int i = 0; i < ARRAY_SIZE; i++)
		in[i] = rand() % 10;

	// Single stram version
	timer.onTimer(0);

	timer.onTimer(1);
	hipMemcpy(dIn, in, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice);
	timer.offTimer(1);

	timer.onTimer(2);
	myKernel <<<NUM_BLOCK, 1024>>> (dIn, dOut);
	hipDeviceSynchronize();
	timer.offTimer(2);

	timer.onTimer(3);
	hipMemcpy(out, dOut, sizeof(int) * ARRAY_SIZE, hipMemcpyDeviceToHost);
	timer.offTimer(3);

	timer.offTimer(0);

	// Multiple stream version
	hipStream_t stream[NUM_STREAMS];

	for (int i = 0; i < NUM_STREAMS; i++)
		hipStreamCreate(&stream[i]);

	int chunkSize = ARRAY_SIZE / NUM_STREAMS;

	timer.onTimer(4);
	for (int i = 0; i < NUM_STREAMS; i++)
	{
		int offset = chunkSize * i;
		hipMemcpyAsync(dIn + offset, in + offset, sizeof(int) * chunkSize, hipMemcpyHostToDevice, stream[i]);
		myKernel <<<NUM_BLOCK / NUM_STREAMS, 1024, 0, stream[i] >> > (dIn + offset, dOut + offset);
		hipMemcpyAsync(out2 + offset, dOut + offset, sizeof(int) * chunkSize, hipMemcpyDeviceToHost, stream[i]);
	}

	hipDeviceSynchronize();
	timer.offTimer(4);

	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		if (out[i] != out2[i])
			printf("!");
	}

	for (int i = 0; i < NUM_STREAMS; i++)
		hipStreamDestroy(stream[i]);

	timer.printTimer();

	hipFree(dIn);
	hipFree(dOut);

	hipHostFree(in);
	hipHostFree(out);
	hipHostFree(out2);
}