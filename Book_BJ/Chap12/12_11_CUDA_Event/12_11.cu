#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ARRAY_SIZE (64*1024*1024)

#define NUM_STREAMS 4

__global__ void myKernel(int* _in, int* _out)
{
	int tID = blockDim.x * blockIdx.x + threadIdx.x;

	int temp = 0;
	for (int i = 0; i < 500; i++) {
		temp = (temp + _in[tID] * 5) % 10;
	}
	_out[tID] = temp;

}

void main(void)
{
	DS_timer timer(1);
	timer.setTimerName(0, "Total");

	int* in = NULL, * out = NULL;

	hipHostMalloc(&in, sizeof(int) * ARRAY_SIZE);
	memset(in, 0, sizeof(int) * ARRAY_SIZE);

	hipHostMalloc(&out, sizeof(int) * ARRAY_SIZE);
	memset(out, 0, sizeof(int) * ARRAY_SIZE);

	int* dIn, * dOut;
	hipMalloc(&dIn, sizeof(int) * ARRAY_SIZE);
	hipMalloc(&dOut, sizeof(int) * ARRAY_SIZE);

	for (int i = 0; i < ARRAY_SIZE; i++)
		in[i] = rand() % 1000;

	hipStream_t stream[NUM_STREAMS];
	hipEvent_t start[NUM_STREAMS], end[NUM_STREAMS];

	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&stream[i]);
		hipEventCreate(&start[i]); hipEventCreate(&end[i]);
	}

	int chunkSize = ARRAY_SIZE / NUM_STREAMS;

	int offset[NUM_STREAMS] = { 0 };
	for (int i = 0; i < NUM_STREAMS; i++)
		offset[i] = chunkSize * i;

	timer.onTimer(0);

	for (int i = 0; i < NUM_STREAMS; i++) {
		hipEventRecord(start[i], stream[i]);
		hipMemcpyAsync(dIn + offset[i], in + offset[i], sizeof(int) * chunkSize, hipMemcpyHostToDevice, stream[i]);
	}

	for (int i = 0; i < NUM_STREAMS; i++)
		myKernel <<<chunkSize / 1024, 1024, 0, stream[i] >> > (dIn + offset[i], dOut + offset[i]);

	for (int i = 0; i < NUM_STREAMS; i++) {
		hipMemcpyAsync(out + offset[i], dOut + offset[i], sizeof(int) * chunkSize, hipMemcpyDeviceToHost, stream[i]);
		hipEventRecord(end[i], stream[i]);
	}

	hipDeviceSynchronize();
	timer.offTimer(0);
	timer.printTimer();

	for (int i = 0; i < NUM_STREAMS; i++) {
		float time = 0;
		hipEventElapsedTime(&time, start[i], end[i]);
		printf("Stream[%d] : %f ms\n", i, time);
	}

	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamDestroy(stream[i]);
		hipEventDestroy(start[i]);
		hipEventDestroy(end[i]);
	}

	hipFree(dIn);
	hipFree(dOut);

	hipHostFree(in);
	hipHostFree(out);
}