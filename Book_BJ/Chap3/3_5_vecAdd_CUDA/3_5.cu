﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// The size of the vector
#define NUM_DATA 1024

// Simple vector sum kernel (Max vector size : 1024)
__global__ void vecAdd(int* _a, int* _b, int* _c) {
	int tID = threadIdx.x;
	_c[tID] = _a[tID] + _b[tID];
}

int main(void)
{
	int* a, * b, * c, * hc;	// Vectors on the host
	int* da, * db, * dc;	// Vectors on the device

	int memSize = sizeof(int) * NUM_DATA;
	printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

	// Memory allocation on the host-side
	a = new int[NUM_DATA]; memset(a, 0, memSize);
	b = new int[NUM_DATA]; memset(b, 0, memSize);
	c = new int[NUM_DATA]; memset(c, 0, memSize);
	hc = new int[NUM_DATA]; memset(hc, 0, memSize);

	// Data generation
	for (int i = 0; i < NUM_DATA; i++) {
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	// Vector sum on host (for performance comparision)
	for (int i = 0; i < NUM_DATA; i++)
		hc[i] = a[i] + b[i];

	// Memory allocation on the device-side
	hipMalloc(&da, memSize); hipMemset(da, 0, memSize);
	hipMalloc(&db, memSize); hipMemset(db, 0, memSize);
	hipMalloc(&dc, memSize); hipMemset(dc, 0, memSize);

	// Data copy : Host -> Device
	hipMemcpy(da, a, memSize, hipMemcpyHostToDevice);
	hipMemcpy(db, b, memSize, hipMemcpyHostToDevice);

	// Kernel call
	vecAdd <<<1, NUM_DATA >>> (da, db, dc);

	// Copy results : Device -> Host
	hipMemcpy(c, dc, memSize, hipMemcpyDeviceToHost);

	// Release device memory
	hipFree(da); hipFree(db); hipFree(dc);

	// Check results
	bool result = true;
	for (int i = 0; i < NUM_DATA; i++) {
		if (hc[i] != c[i]) {
			printf("[%d] The result is not matched! (%d, %d)\n"
				, i, hc[i], c[i]);
			result = false;
		}
	}

	if (result)
		printf("GPU works well!\n");

	// Release host memory
	delete[] a; delete[] b; delete[] c;

	return 0;
}