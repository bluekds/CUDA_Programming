#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void)
{
	size_t free, total;
	hipMemGetInfo(&free, &total); // 현재 사용가능한 device memory 크기와, 총 device 메모리 크기를 얻어오는 함수
	printf("Device memory (free/total) = %lld/%lld bytes\n"
		, free, total);
}

int main(void)
{
	int* dDataPtr;
	hipError_t errorCode ;

	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024); // device memory 할당
	printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024); // device memory 초기화
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));

	errorCode = hipFree(dDataPtr);
	printf("hipFree - %s\n", hipGetErrorName(errorCode)); // device memory 해제
	checkDeviceMemory();
}