#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_CPU_THREADS	(4)

#define ROW_SIZE (32)
#define K_SIZE   (128)
#define COL_SIZE (32)

#define MAT_SIZE_A (ROW_SIZE*K_SIZE)
#define MAT_SIZE_B (K_SIZE*COL_SIZE)
#define MAT_SIZE_C (ROW_SIZE*COL_SIZE)

// input matrix
float A[ROW_SIZE][K_SIZE];	// m * k
float B[K_SIZE][COL_SIZE];	// k * n

// timer
DS_timer* timer;
void setTimer(void);
#define TIMER_HOST			0
#define TIMER_KERNEL		1
#define TIMER_KERNEL_SH		2
#define TIMER_KERNEL_SH_C	3
#define TIMER_HtoD			4
#define TIMER_DtoH			5
#define NUM_TIMER			(TIMER_DtoH+1)

void genInputMatrices(void);

// output matrix
float hostC[ROW_SIZE][COL_SIZE];	// host result
float deviceC[COL_SIZE][COL_SIZE];	// device result

#define memsetZero(_P,_type,_size) memset(_P, 0, sizeof(_type)*_size);
#define dMemAlloc(_P, _type, _size) hipMalloc(&_P, sizeof(_type)*_size);


__global__ void matMul_kernel(float* _A, float* _B, float* _C)
{
	int row = threadIdx.x;
	int col = threadIdx.y;
	int index = row * blockDim.y + col;

	float result = 0;
	for (int k = 0; k < K_SIZE; k++)
		result += _A[row * K_SIZE + k] * _B[col + k * COL_SIZE];
	_C[index] = result;
}

__global__ void matMul_kernel_shared(float* _A, float* _B, float* _C)
{
	int row = threadIdx.x;
	int col = threadIdx.y;
	int index = row * blockDim.y + col;

	__shared__ float sA[ROW_SIZE][K_SIZE];	// 32 * 256 * 4 bytes = 16 KB
	__shared__ float sB[K_SIZE][COL_SIZE];	// 16 KB

	if (row == 0) { // read matrix B
		for (int k = 0; k < K_SIZE; k++)
			sB[k][col] = _B[col + k * COL_SIZE];
	}
	else if (col == 0 ) { // read matrix A
		for (int k = 0; k < K_SIZE; k++)
			sA[row][k] = _A[row * K_SIZE + k];

	}
	if (row == 0 && col == 0) { // read the first row of A
		for (int k = 0; k < K_SIZE; k++)
			sA[row][k] = _A[row * K_SIZE + k];
	}

	__syncthreads(); // wait until all thread load the matrix

	float result = 0;
	for (int k = 0; k < K_SIZE; k++)
		result += sA[row][k] * sB[k][col];
	_C[index] = result;
}

void main(void)
{
	timer = NULL;	setTimer();

	float* dA, * dB, * dC;
	dA = dB = dC = NULL;

	memsetZero(A, float, MAT_SIZE_A);	memsetZero(B, float, MAT_SIZE_B);
	memsetZero(hostC, float, MAT_SIZE_C);	memsetZero(deviceC, float, MAT_SIZE_C);

	// device memory allocaiton
	dMemAlloc(dA, float, MAT_SIZE_A);
	dMemAlloc(dB, float, MAT_SIZE_B);
	dMemAlloc(dC, float, MAT_SIZE_C);

	// generate input matrices
	genInputMatrices();

	// Host code
	timer->onTimer(TIMER_HOST);
	for (int r = 0; r < ROW_SIZE; r++)
		for (int c = 0; c < COL_SIZE; c++)
			for (int k = 0; k < K_SIZE; k++)
				hostC[r][c] += A[r][k] * B[k][c];
	timer->offTimer(TIMER_HOST);

	// Copy input matrices : H -> D
	timer->onTimer(TIMER_HtoD);
	hipMemcpy(dA, A, sizeof(float) * MAT_SIZE_A, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(float) * MAT_SIZE_B, hipMemcpyHostToDevice);
	timer->offTimer(TIMER_HtoD);

	dim3 blockDim(ROW_SIZE, COL_SIZE);

	timer->onTimer(TIMER_KERNEL);
	matMul_kernel << <1, blockDim >> > (dA, dB, dC);
	hipDeviceSynchronize();
	timer->offTimer(TIMER_KERNEL);

	timer->onTimer(TIMER_KERNEL_SH);
	matMul_kernel_shared << <1, blockDim >> > (dA, dB, dC);
	hipDeviceSynchronize();
	timer->offTimer(TIMER_KERNEL_SH);

	// Get back result : D -> H
	timer->onTimer(TIMER_DtoH);
	hipMemcpy(deviceC, dC, sizeof(float) * MAT_SIZE_C, hipMemcpyDeviceToHost);
	timer->onTimer(TIMER_DtoH);

	// check the results
	bool isCorrect = true;

	float* pHostC = &hostC[0][0];
	float* pDeviceC = &deviceC[0][0];

	for (int i = 0; i < MAT_SIZE_C; i++) {
		if (pHostC[i] != pDeviceC[i]) {
			printf("[%d] %.2f, %.2f\n", i, pHostC[i], pDeviceC[i]);
			isCorrect = false;
			break;
		}
	}

	if (isCorrect)
		printf("Result is correct!\n");
	else
		printf("Result is not correct!!!!!!\n");

	timer->printTimer();
	if (timer != NULL)
		delete timer;
}

void genInputMatrices(void)
{
	for (int r = 0; r < ROW_SIZE; r++)
		for (int k = 0; k < K_SIZE; k++)
			A[r][k] = rand() % 100;

	for (int k = 0; k < K_SIZE; k++)
		for (int c = 0; c < COL_SIZE; c++)
			B[k][c] = rand() % 100;
}

void setTimer(void)
{
	timer = new DS_timer(NUM_TIMER);

	timer->initTimers();
	timer->setTimerName(TIMER_HOST, (char*)"CPU code");
	timer->setTimerName(TIMER_KERNEL, (char*)"Kernel launch");
	timer->setTimerName(TIMER_KERNEL_SH, (char*)"Kernel launch (shared ver.)");
	timer->setTimerName(TIMER_HtoD, (char*)"[Data transter] host->device");
	timer->setTimerName(TIMER_DtoH, (char*)"[Data transfer] device->host");
}